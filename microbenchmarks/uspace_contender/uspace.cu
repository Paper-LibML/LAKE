#include <sys/mman.h>
#include <stdint.h>
#include <unistd.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <jhash.h>
#include <chrono>
#include <iostream>

#define PAGE_SIZE 4096 // getpagesize()

int main(int argc, char** argv) {
  int res;
  uint64_t seed = 17;
  uint64_t max_concurrency = 1;
	uint64_t batch_size = 4096;
  uint64_t n_pages = 128 * batch_size;
  uint64_t grid_x = 4;
  uint64_t grid_y = 4;
  uint64_t grid_z = 4;
  uint64_t block_z = 1;
  uint64_t block_y = 1;
  uint64_t block_x = batch_size / (grid_x * grid_y * grid_z * block_y * block_z);

  // CUDA setup
  res = hipInit(0);
  if (res) { printf("Error cuinit (%d)\n", res); }
  hipDevice_t cuDevice;
  res = hipDeviceGet(&cuDevice, 0);
  if (res) { printf("Error getDev (%d)\n", res); }
  hipCtx_t cuContext;
  res = hipCtxCreate(&cuContext, 0, cuDevice);
  if (res) { printf("Error ctxCreat (%d)\n", res); }
  hipModule_t cuModule;
  res = hipModuleLoad(&cuModule, "jhash.cubin");
  if (res) { printf("Error loadMod (%d), %s\n", res, hipGetErrorString((hipError_t)res)); }
  hipStream_t streams[max_concurrency];
  for (int i = 0; i < max_concurrency; ++i) {
    res = hipStreamCreateWithFlags(&streams[i], 0);
    if (res) { printf("Error streamCreate (%d), stream no %d\n", res, i); }
  }

  // Create a model identical page
  char *page = (char *) malloc(PAGE_SIZE);
  for (int i = 0; i < PAGE_SIZE; ++i) {
    page[i] = i;
  }

  // mmap pages
  void *pages = mmap(NULL, PAGE_SIZE * n_pages,
    PROT_READ | PROT_WRITE, MAP_ANONYMOUS | MAP_PRIVATE, -1, 0);
  for (uint64_t i = 0; i < n_pages; ++i) {
    char *page_start = ((char *) pages) + PAGE_SIZE * i;
    memcpy(page_start, page, PAGE_SIZE);
  }

  // alloc checksum buf
  uint32_t *h_checksum = (uint32_t *) malloc(batch_size * max_concurrency * sizeof(uint32_t));

  // alloc zero-copy pinned memory
  void *h_pages;
  res = hipHostMalloc(&h_pages, batch_size * PAGE_SIZE * max_concurrency);
//  h_pages = malloc(batch_size * PAGE_SIZE * max_concurrency);
  if (res) { printf("Couldn't allocate pinned pages (%d)\n", res); }

  // Device side mem
  hipDeviceptr_t d_pages;
  hipDeviceptr_t d_checksum;
  res = hipMalloc(&d_pages, batch_size * PAGE_SIZE * max_concurrency);
  if (res) { printf("Error memalloc 1 (%d)\n", res); }
  res = hipMalloc(&d_checksum, batch_size * max_concurrency * sizeof(uint32_t));
  if (res) { printf("Error memalloc 2 (%d)\n", res); }

  hipFunction_t xxh;
  res = hipModuleGetFunction(&xxh, cuModule, "_Z6jhash2PvPj");
  if (res != hipSuccess) { printf("Error getting function (%d)\n", res); }

//  // ======================= Test only memcpy tpt ===========================
//
//  double total_cpu_cpu_memcpy_time = 0.0;
//  for (uint64_t i = 0; i < n_pages; i += batch_size * max_concurrency) {
//    for (int j = 0; j < max_concurrency; ++j) {
//      std::chrono::high_resolution_clock::time_point t5 =
//        std::chrono::high_resolution_clock::now();
//
//      memcpy(((char *) h_pages) + j * batch_size * PAGE_SIZE, ((char *) pages) + (i + j * batch_size) * PAGE_SIZE, batch_size * PAGE_SIZE);
//
//      std::chrono::high_resolution_clock::time_point t6 =
//        std::chrono::high_resolution_clock::now();
//      total_cpu_cpu_memcpy_time +=
//        std::chrono::duration_cast<std::chrono::duration<double>>(t6 - t5).count();
//    }
//  }

  // ============================ GPU checksum ============================

  // Measure tpt
  uint64_t n_samples = 5000;
  int n_iterations = 500;
  uint64_t batches_per_sample = n_iterations * n_pages * max_concurrency / (n_samples * batch_size);
  uint64_t *pages_checksummed = (uint64_t *) malloc(n_samples * sizeof(uint64_t));
  std::chrono::high_resolution_clock::time_point *times = (std::chrono::high_resolution_clock::time_point *) malloc(n_samples * sizeof(std::chrono::high_resolution_clock::time_point));

  double total_memcpy_time = 0.0;
  // Time start
  std::chrono::high_resolution_clock::time_point t1 =
    std::chrono::high_resolution_clock::now();

  int j = 0;
  uint64_t i = 0;
  int sample_cnt = 0;


  // Run all the batches on GPU
  for (int k = 0; k < n_iterations; ++k) {
    for (i = 0; i < n_pages; i += batch_size * max_concurrency) {
      for (j = 0; j < max_concurrency; ++j) {
    hipDeviceptr_t concur_pages = d_pages + j * batch_size * PAGE_SIZE;
    hipDeviceptr_t concur_checksum = d_checksum + j * batch_size * sizeof(uint32_t);

    if (j == 0 && k == 0) {
      memcpy(((char *) h_pages) + j * batch_size * PAGE_SIZE, ((char *) pages) + (i + j * batch_size) * PAGE_SIZE, batch_size * PAGE_SIZE);
  
  
      // Copy to dev
      res = hipMemcpyHtoDAsync(concur_pages, h_pages, batch_size * PAGE_SIZE, streams[j]);
      if (res) { printf("Error memcpy htod 1 (%d)\n", res); }
      hipStreamSynchronize(streams[j]);
      pages_checksummed[0] = 0;
      times[0] = std::chrono::high_resolution_clock::now();
    }

        hipStreamSynchronize(streams[j]);

        // Launch kernel
        void *args[] = { &concur_pages, &concur_checksum };
        res = hipModuleLaunchKernel(xxh, grid_x, grid_y, grid_z, block_x, block_y, block_z,
          0, streams[j], args, NULL);
        if (res) { printf("Error launching kernel (%d)\n", res); }

    if (j == 0 && k == 0) {
      // Copy to host
      res = hipMemcpyDtoHAsync(h_checksum + j * batch_size, concur_checksum,
        batch_size * sizeof(uint32_t), streams[0]);
      if (res) { printf("Error memcpy dtoh (%d)\n", res); }
    }


//   // TODO: remove
//   // Check results
//    uint32_t checksum1 = jhash2((uint32_t *) pages, PAGE_SIZE / sizeof(uint32_t), seed);
//    for (int x = 0; x < batch_size * max_concurrency; ++x) {
//      if (checksum1 != h_checksum[x]) {
//        printf("Checksums don't match, idx = %d, should be %x, is %x\n", x,
//          checksum1, h_checksum[x]);
//      }
//    }
//  	return 0;
//   // TODO: /remove

        if ((k * n_pages + i) / batch_size % batches_per_sample == 0 && sample_cnt < n_samples - 1 ) {
          pages_checksummed[sample_cnt + 1] = pages_checksummed[sample_cnt] + batches_per_sample * batch_size;
          times[sample_cnt + 1] = std::chrono::high_resolution_clock::now();
          sample_cnt++;
        }
      }
    }
  }

  // Time end
  std::chrono::high_resolution_clock::time_point t2 =
    std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> gpu_time =
    std::chrono::duration_cast<std::chrono::duration<double>>(t2 - t1);

  // ============================ End GPU checksum ============================

  // Check results
  uint32_t checksum = jhash2((uint32_t *) pages, PAGE_SIZE / sizeof(uint32_t), seed);
  for (int i = 0; i < batch_size * max_concurrency; ++i) {
    if (checksum != h_checksum[i]) {
      printf("Checksums don't match, idx = %d, should be %x, is %x\n", i,
        checksum, h_checksum[i]);
    }
  }

  // ============================ CPU checksum ============================
  uint32_t * host_checksum_vals = (uint32_t *) malloc(n_pages * PAGE_SIZE);

  // Time start
  std::chrono::high_resolution_clock::time_point t3 =
    std::chrono::high_resolution_clock::now();

  // hash all the pages on cpu
  for (int i = 0; i < n_pages; ++i) {
    uint32_t *h_page = ((uint32_t *) pages) + i / sizeof(uint32_t);
    host_checksum_vals[i] = jhash2(h_page, PAGE_SIZE, seed);
  }

  // Time end
  std::chrono::high_resolution_clock::time_point t4 =
    std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> cpu_time =
    std::chrono::duration_cast<std::chrono::duration<double>>(t4 - t3);

  // ============================ End CPU checksum ============================

  // Print speedup
//  printf("CPU time: %0.4f sec, GPU time: %0.4f sec\n", cpu_time.count(), gpu_time.count());
//  printf("Speedup: %0.3fx\n",
//    cpu_time.count() / gpu_time.count());
//  printf("Memcpy throughput: %0.3f GB / sec\n", ((double) n_pages * PAGE_SIZE) / total_memcpy_time / (1024 * 1024 * 1024));
  hipHostFree(h_pages);
 
//  pages_checksummed[0] = 0;
//  times[0] = std::chrono::high_resolution_clock::now();

  // Print tpt
  printf("time,tpt\n");
  for (int l = 1; l < n_samples; ++l) {
    printf("%f,%f\n", std::chrono::duration_cast<std::chrono::duration<double>>(times[l] - times[0]).count(), (double) (pages_checksummed[l] - pages_checksummed[l - 1]) / (std::chrono::duration_cast<std::chrono::duration<double>>(times[l] - times[l - 1]).count()));
  }
}
